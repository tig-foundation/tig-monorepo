/*!
Copyright 2025 Rootz

Identity of Submitter Rootz

UAI null

Licensed under the TIG Inbound Game License v2.0 or (at your option) any later
version (the "License"); you may not use this file except in compliance with the
License. You may obtain a copy of the License at

https://github.com/tig-foundation/tig-monorepo/tree/main/docs/licenses

Unless required by applicable law or agreed to in writing, software distributed
under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
CONDITIONS OF ANY KIND, either express or implied. See the License for the specific
language governing permissions and limitations under the License.
*/
#include <stdint.h>
#include <hip/hip_runtime.h>

extern "C" __global__ void hyperedge_clustering(
    const int num_hyperedges,
    const int num_clusters,
    const int * __restrict__ hyperedge_offsets,
    int * __restrict__ hyperedge_clusters
) {
    int hedge = blockIdx.x * blockDim.x + threadIdx.x;

    if (hedge < num_hyperedges) {
        int start = hyperedge_offsets[hedge];
        int end = hyperedge_offsets[hedge + 1];
        int hedge_size = end - start;

        int quarter_clusters = max(1, num_clusters >> 2);
        int cluster_mask = quarter_clusters - 1;

        int cluster;
        if (hedge_size <= 2) {
            cluster = hedge & cluster_mask;
        } else if (hedge_size <= 4) {
            cluster = quarter_clusters + (hedge & cluster_mask);
        } else if (hedge_size <= 8) {
            cluster = (quarter_clusters << 1) + (hedge & cluster_mask);
        } else {
            cluster = (quarter_clusters * 3) + (hedge & cluster_mask);
        }

        hyperedge_clusters[hedge] = cluster;
    }
}

extern "C" __global__ void compute_node_preferences(
    const int num_nodes,
    const int num_parts,
    const int num_hedge_clusters,
    const int * __restrict__ node_hyperedges,
    const int * __restrict__ node_offsets,
    const int * __restrict__ hyperedge_clusters,
    const int * __restrict__ hyperedge_offsets,
    int * __restrict__ pref_parts,
    int * __restrict__ pref_priorities
) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;

    if (node < num_nodes) {
        int start = node_offsets[node];
        int end = node_offsets[node + 1];
        int node_degree = end - start;

        unsigned short cluster_votes[256];
        int max_clusters = min(num_hedge_clusters, 256);
        for (int i = 0; i < max_clusters; i++) {
            cluster_votes[i] = 0;
        }

        int max_votes = 0;
        int best_cluster = 0;

        for (int j = start; j < end; j++) {
            int hyperedge = node_hyperedges[j];
            int cluster = hyperedge_clusters[hyperedge];

            if (cluster >= 0 && cluster < max_clusters) {
                int hedge_start = hyperedge_offsets[hyperedge];
                int hedge_end = hyperedge_offsets[hyperedge + 1];
                int hedge_size = hedge_end - hedge_start;
                int weight = (hedge_size <= 3) ? 3 : (hedge_size <= 6) ? 2 : 1;

                cluster_votes[cluster] += weight;

                if (cluster_votes[cluster] > max_votes ||
                    (cluster_votes[cluster] == max_votes && cluster < best_cluster)) {
                    max_votes = cluster_votes[cluster];
                    best_cluster = cluster;
                }
            }
        }

        int target_partition;
        if (node_degree <= 3) {
            target_partition = (best_cluster + node) % num_parts;
        } else if (node_degree <= 8) {
            target_partition = (best_cluster + node_degree + node) % num_parts;
        } else {
            target_partition = (best_cluster * 2 + node_degree + node) % num_parts;
        }

        pref_parts[node] = target_partition;
        pref_priorities[node] = (max_votes << 16) + (num_parts - (node % num_parts));
    }
}

extern "C" __global__ void execute_node_assignments(
    const int num_nodes,
    const int num_parts,
    const int max_part_size,
    const int *sorted_nodes,
    const int *sorted_parts,
    int *partition,
    int *nodes_in_part
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < num_nodes; i++) {
            int node = sorted_nodes[i];
            int preferred_part = sorted_parts[i];

            if (node >= 0 && node < num_nodes && preferred_part >= 0 && preferred_part < num_parts) {
                bool assigned = false;
                for (int attempt = 0; attempt < num_parts; attempt++) {
                    int try_part = (preferred_part + attempt) % num_parts;
                    if (nodes_in_part[try_part] < max_part_size) {
                        partition[node] = try_part;
                        nodes_in_part[try_part]++;
                        assigned = true;
                        break;
                    }
                }

                if (!assigned) {
                    int fallback_part = node % num_parts;
                    partition[node] = fallback_part;
                    nodes_in_part[fallback_part]++;
                }
            }
        }
    }
}

extern "C" __global__ void compute_refinement_moves_batched(
    const int batch_start,
    const int batch_size,
    const int num_nodes,
    const int num_parts,
    const int max_part_size,
    const int num_hyperedges,
    const int * __restrict__ node_hyperedges,
    const int * __restrict__ node_offsets,
    const int * __restrict__ hyperedge_nodes,
    const int * __restrict__ hyperedge_offsets,
    const int * __restrict__ partition,
    const int * __restrict__ nodes_in_part,
    int * __restrict__ move_parts,
    int * __restrict__ move_gains,
    int * __restrict__ move_priorities,
    int * __restrict__ num_valid_moves,
    const int round
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int node = batch_start + idx;
    if (idx >= batch_size || node >= num_nodes) return;

    move_parts[node] = partition[node];
    move_gains[node] = 0;
    move_priorities[node] = 0;

    int current_part = partition[node];
    if (current_part < 0 || current_part >= num_parts || nodes_in_part[current_part] <= 1) return;

    int start = node_offsets[node];
    int end = node_offsets[node + 1];
    int node_degree = end - start;
    if (node_degree <= 0) return;

    const int MAX_FLAGS = 64;
    int eff_degree = (node_degree < MAX_FLAGS) ? node_degree : MAX_FLAGS;
    unsigned long long edge_flags[MAX_FLAGS];
    int max_p = min(num_parts, 64);

    #pragma unroll
    for (int j = 0; j < eff_degree; j++) {
        edge_flags[j] = 0ULL;
        int hyperedge = node_hyperedges[start + j];
        int hedge_start = hyperedge_offsets[hyperedge];
        int hedge_end = hyperedge_offsets[hyperedge + 1];

        #pragma unroll 4
        for (int k = hedge_start; k < hedge_end; k++) {
            int other_node = hyperedge_nodes[k];
            if (other_node != node && other_node >= 0 && other_node < num_nodes) {
                int part = partition[other_node];
                if (part >= 0 && part < max_p) {
                    edge_flags[j] |= 1ULL << (part & 63);
                }
            }
        }
    }

    int cp = current_part & 63;
    int original_cost = 0;
    #pragma unroll 4
    for (int j = 0; j < eff_degree; j++) {
        int lambda = __popcll(edge_flags[j] | (1ULL << cp));
        if (lambda > 1) original_cost += (lambda - 1);
    }

    int best_gain = 0;
    int best_target = current_part;
    int current_size = nodes_in_part[current_part];
    int balance_threshold = (num_hyperedges < 50000) ? 2 : 4;

    for (int offset = 0; offset < num_parts; offset++) {
        int target_part = (node + round + offset) % num_parts;
        if (target_part == current_part) continue;

        int target_size = nodes_in_part[target_part];
        if (target_size >= max_part_size) continue;

        int tp = target_part & 63;
        int new_cost = 0;
        #pragma unroll 8
        for (int j = 0; j < eff_degree; j++) {
            int lambda = __popcll(edge_flags[j] | (1ULL << tp));
            if (lambda > 1) new_cost += (lambda - 1);
        }

        int basic_gain = original_cost - new_cost;
        int balance_bonus = (current_size > target_size + 2) ? balance_threshold : 0;
        int total_gain = basic_gain + balance_bonus;

        if (total_gain > best_gain || (total_gain == best_gain && target_part < best_target)) {
            best_gain = total_gain;
            best_target = target_part;
        }
    }

    if (best_gain > 0 && best_target != current_part) {
        move_parts[node] = best_target;
        move_gains[node] = best_gain;
        move_priorities[node] = (best_gain << 16) + (num_parts - (node % num_parts));
        atomicAdd(num_valid_moves, 1);
    }
}

extern "C" __global__ void execute_refinement_moves(
    const int num_valid_moves,
    const int * __restrict__ sorted_nodes,
    const int * __restrict__ sorted_parts,
    const int max_part_size,
    int * __restrict__ partition,
    int * __restrict__ nodes_in_part,
    int * __restrict__ moves_executed
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < num_valid_moves; i++) {
            int node = sorted_nodes[i];
            int target_part = sorted_parts[i];

            if (node >= 0 && target_part >= 0) {
                int current_part = partition[node];

                if (current_part >= 0 &&
                    nodes_in_part[target_part] < max_part_size &&
                    nodes_in_part[current_part] > 1 &&
                    partition[node] == current_part) {

                    partition[node] = target_part;
                    nodes_in_part[current_part]--;
                    nodes_in_part[target_part]++;
                    (*moves_executed)++;
                }
            }
        }
    }
}

extern "C" __global__ void balance_final(
    const int num_nodes,
    const int num_parts,
    const int min_part_size,
    const int max_part_size,
    int *partition,
    int *nodes_in_part
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int part = 0; part < num_parts; part++) {
            while (nodes_in_part[part] < min_part_size) {
                bool moved = false;
                for (int other_part = 0; other_part < num_parts && !moved; other_part++) {
                    if (other_part != part && nodes_in_part[other_part] > min_part_size) {
                        for (int node = 0; node < num_nodes; node++) {
                            if (partition[node] == other_part) {
                                partition[node] = part;
                                nodes_in_part[other_part]--;
                                nodes_in_part[part]++;
                                moved = true;
                                break;
                            }
                        }
                    }
                }
                if (!moved) break;
            }
        }

        for (int part = 0; part < num_parts; part++) {
            while (nodes_in_part[part] > max_part_size) {
                bool moved = false;
                for (int other_part = 0; other_part < num_parts && !moved; other_part++) {
                    if (other_part != part && nodes_in_part[other_part] < max_part_size) {
                        for (int node = 0; node < num_nodes; node++) {
                            if (partition[node] == part) {
                                partition[node] = other_part;
                                nodes_in_part[part]--;
                                nodes_in_part[other_part]++;
                                moved = true;
                                break;
                            }
                        }
                    }
                }
                if (!moved) break;
            }
        }
    }
}
