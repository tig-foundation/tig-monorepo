/*!Copyright 2025 Rootz

Identity of Submitter Rootz

UAI null

Licensed under the TIG Inbound Game License v2.0 or (at your option) any later
version (the "License"); you may not use this file except in compliance with the
License. You may obtain a copy of the License at

https://github.com/tig-foundation/tig-monorepo/tree/main/docs/licenses

Unless required by applicable law or agreed to in writing, software distributed
under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
CONDITIONS OF ANY KIND, either express or implied. See the License for the specific
language governing permissions and limitations under the License.
*/
#include <hip/hip_runtime.h>
#include <float.h>

#define MAX_FLOAT 3.402823466e+38F

__device__ __forceinline__ float euclidean_distance(const float* __restrict__ a, const float* __restrict__ b, int dims) {
    float sum = 0.0f;
    float c = 0.0f;
    int i;
    
    for (i = 0; i < dims - 15; i += 16) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];

        float s0 = d0*d0 + d1*d1 + d2*d2 + d3*d3;
        float s1 = d4*d4 + d5*d5 + d6*d6 + d7*d7;
        float s2 = d8*d8 + d9*d9 + d10*d10 + d11*d11;
        float s3 = d12*d12 + d13*d13 + d14*d14 + d15*d15;

        float partial = s0 + s1 + s2 + s3;
        float y = partial - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    for (; i < dims - 7; i += 8) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims - 3; i += 4) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims; i++) {
        float diff = a[i] - b[i];
        float squared = diff * diff;
        float y = squared - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    return sum;
}

__device__ __forceinline__ float euclidean_distance_high(const float* __restrict__ a, const float* __restrict__ b, int dims) {
    float sum = 0.0f;
    float c = 0.0f;
    int i;

    for (i = 0; i < dims - 31; i += 32) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];
        float d16=a[i+16]-b[i+16], d17=a[i+17]-b[i+17], d18=a[i+18]-b[i+18], d19=a[i+19]-b[i+19];
        float d20=a[i+20]-b[i+20], d21=a[i+21]-b[i+21], d22=a[i+22]-b[i+22], d23=a[i+23]-b[i+23];
        float d24=a[i+24]-b[i+24], d25=a[i+25]-b[i+25], d26=a[i+26]-b[i+26], d27=a[i+27]-b[i+27];
        float d28=a[i+28]-b[i+28], d29=a[i+29]-b[i+29], d30=a[i+30]-b[i+30], d31=a[i+31]-b[i+31];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d8*d8; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d9*d9; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d10*d10; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d11*d11; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d12*d12; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d13*d13; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d14*d14; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d15*d15; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d16*d16; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d17*d17; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d18*d18; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d19*d19; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d20*d20; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d21*d21; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d22*d22; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d23*d23; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d24*d24; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d25*d25; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d26*d26; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d27*d27; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d28*d28; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d29*d29; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d30*d30; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d31*d31; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims - 15; i += 16) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];

        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d8*d8; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d9*d9; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d10*d10; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d11*d11; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d12*d12; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d13*d13; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d14*d14; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d15*d15; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims - 7; i += 8) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims - 3; i += 4) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
    }

    for (; i < dims; i++) {
        float diff = a[i] - b[i];
        float squared = diff * diff;
        float y = squared - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    return sum;
}

__device__ __forceinline__ float euclidean_distance_bounded(const float* __restrict__ a, const float* __restrict__ b, int dims, float limit) {
    float sum = 0.0f;
    float c = 0.0f;
    float margin = fmaxf(1e-6f, 1.0e-4f * (1.0f + limit));
    int i;
    for (i = 0; i < dims - 15; i += 16) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];
        float s0=d0*d0+d1*d1+d2*d2+d3*d3;
        float s1=d4*d4+d5*d5+d6*d6+d7*d7;
        float s2=d8*d8+d9*d9+d10*d10+d11*d11;
        float s3=d12*d12+d13*d13+d14*d14+d15*d15;
        float partial=s0+s1+s2+s3;
        float y=partial-c;
        float t=sum+y;
        c=(t-sum)-y;
        sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims - 7; i += 8) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims - 3; i += 4) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims; i++) {
        float diff=a[i]-b[i];
        float squared=diff*diff;
        float y=squared-c;
        float t=sum+y;
        c=(t-sum)-y;
        sum=t;
        if (sum > limit + margin) return sum;
    }
    return sum;
}

__device__ __forceinline__ float euclidean_distance_high_bounded(const float* __restrict__ a, const float* __restrict__ b, int dims, float limit) {
    float sum=0.0f;
    float c=0.0f;
    float margin = fmaxf(1e-6f, 1.0e-4f * (1.0f + limit));
    int i;
    for (i=0;i<dims-31;i+=32){
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];
        float d16=a[i+16]-b[i+16], d17=a[i+17]-b[i+17], d18=a[i+18]-b[i+18], d19=a[i+19]-b[i+19];
        float d20=a[i+20]-b[i+20], d21=a[i+21]-b[i+21], d22=a[i+22]-b[i+22], d23=a[i+23]-b[i+23];
        float d24=a[i+24]-b[i+24], d25=a[i+25]-b[i+25], d26=a[i+26]-b[i+26], d27=a[i+27]-b[i+27];
        float d28=a[i+28]-b[i+28], d29=a[i+29]-b[i+29], d30=a[i+30]-b[i+30], d31=a[i+31]-b[i+31];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d8*d8; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d9*d9; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d10*d10; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d11*d11; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d12*d12; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d13*d13; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d14*d14; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d15*d15; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d16*d16; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d17*d17; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d18*d18; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d19*d19; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d20*d20; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d21*d21; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d22*d22; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d23*d23; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d24*d24; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d25*d25; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d26*d26; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d27*d27; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d28*d28; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d29*d29; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d30*d30; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d31*d31; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims - 15; i += 16) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float d8=a[i+8]-b[i+8], d9=a[i+9]-b[i+9], d10=a[i+10]-b[i+10], d11=a[i+11]-b[i+11];
        float d12=a[i+12]-b[i+12], d13=a[i+13]-b[i+13], d14=a[i+14]-b[i+14], d15=a[i+15]-b[i+15];

        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d8*d8; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d9*d9; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d10*d10; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d11*d11; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d12*d12; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d13*d13; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d14*d14; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d15*d15; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims - 7; i += 8) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float d4=a[i+4]-b[i+4], d5=a[i+5]-b[i+5], d6=a[i+6]-b[i+6], d7=a[i+7]-b[i+7];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d4*d4; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d5*d5; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d6*d6; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d7*d7; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims - 3; i += 4) {
        float d0=a[i]-b[i], d1=a[i+1]-b[i+1], d2=a[i+2]-b[i+2], d3=a[i+3]-b[i+3];
        float v,y,t;
        v=d0*d0; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d1*d1; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d2*d2; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        v=d3*d3; y=v-c; t=sum+y; c=(t-sum)-y; sum=t;
        if (sum > limit + margin) return sum;
    }
    for (; i < dims; i++) {
        float diff=a[i]-b[i];
        float squared=diff*diff;
        float y=squared-c;
        float t=sum+y;
        c=(t-sum)-y;
        sum=t;
        if (sum > limit + margin) return sum;
    }
    return sum;
}

__device__ __forceinline__ float euclidean_distance_precise_bounded(const float* __restrict__ a, const float* __restrict__ b, int dims, float limit) {
    double acc = 0.0;
    double lim = (double)limit;
    for (int i = 0; i < dims; i++) {
        double d = (double)a[i] - (double)b[i];
        acc += d * d;
        if (acc > lim) return (float)acc;
    }
    return (float)acc;
}

extern "C" __global__ void deterministic_clustering(
    const float* __restrict__ database_vectors,
    float* __restrict__ cluster_centers,
    int* __restrict__ cluster_assignments,
    int* __restrict__ cluster_sizes,
    int database_size,
    int vector_dims,
    int num_clusters,
    int num_queries
) {
    int cluster_idx = blockIdx.x;
    int tid = threadIdx.x;

    if (cluster_idx >= num_clusters) return;

    long long seed_idx = ((long long)cluster_idx * 982451653LL + 1566083941LL) % (long long)database_size;
    int stride = max(1, database_size / (num_clusters * 37));
    long long start_idx = seed_idx;

    for (int d = tid; d < vector_dims; d += blockDim.x) {
        float acc = 0.0f;
        long long idx = start_idx;
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            int pos = (int)(idx % (long long)database_size);
            acc += database_vectors[pos * vector_dims + d];
            idx += stride;
        }
        cluster_centers[cluster_idx * vector_dims + d] = acc * 0.25f;
    }

    if (tid == 0) {
        cluster_sizes[cluster_idx] = 0;
    }
}

extern "C" __global__ void assign_clusters(
    const float* __restrict__ database_vectors,
    const float* __restrict__ cluster_centers,
    int* __restrict__ cluster_assignments,
    int* __restrict__ cluster_sizes,
    int database_size,
    int vector_dims,
    int num_clusters,
    int num_queries
) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const bool use_high = (num_queries > 3000) || (vector_dims >= 700);
    if (thread_id < database_size) {
        int vec_idx = thread_id;
        const float* vector = database_vectors + vec_idx * vector_dims;
        float min_dist = MAX_FLOAT;
        int best_cluster = 0;
        for (int c = 0; c < num_clusters; c++) {
            const float* c_center = cluster_centers + c * vector_dims;
            float dist = use_high ? euclidean_distance_high(vector, c_center, vector_dims)
                                  : euclidean_distance(vector, c_center, vector_dims);
            if (dist < min_dist) {
                min_dist = dist;
                best_cluster = c;
            }
        }
        cluster_assignments[vec_idx] = best_cluster;
    }
}

extern "C" __global__ void exclusive_scan_sizes(
    const int* cluster_sizes,
    int* cluster_offsets,
    int* write_offsets,
    int num_clusters
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        int acc = 0;
        for (int i = 0; i < num_clusters; i++) {
            cluster_offsets[i] = acc;
            write_offsets[i] = acc;
            acc += cluster_sizes[i];
        }
    }
}

extern "C" __global__ void build_cluster_index(
    const int* cluster_assignments,
    int* write_offsets,
    int* cluster_indices,
    int database_size
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int vec_idx = 0; vec_idx < database_size; vec_idx++) {
            int cluster = cluster_assignments[vec_idx];
            int pos = write_offsets[cluster];
            cluster_indices[pos] = vec_idx;
            write_offsets[cluster]++;
        }
    }
}

extern "C" __global__ void count_block_cluster_sizes(
    const int* cluster_assignments,
    int* block_counts,
    int database_size,
    int num_clusters
) {
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int block = blockIdx.x;
    int base = block * blockDim.x;
    int vec_idx = base + tid;
    __shared__ int s_len;
    if (tid == 0) {
        int rem = database_size - base;
        s_len = rem > blockDim.x ? blockDim.x : (rem > 0 ? rem : 0);
    }
    __syncthreads();
    if (s_len == 0) {
        if (tid == 0) {
            for (int c = 0; c < num_clusters; c++) {
                block_counts[block * num_clusters + c] = 0;
            }
        }
        return;
    }
    int cid = -1;
    if (tid < s_len) cid = cluster_assignments[vec_idx];

    for (int c = 0; c < num_clusters; c++) {
        int* buf = sdata + c * blockDim.x;
        if (tid < s_len) {
            buf[tid] = (cid == c) ? 1 : 0;
        } else if (tid < blockDim.x) {
            buf[tid] = 0;
        }
    }
    __syncthreads();

    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            int limit = (tid + stride < s_len) ? 1 : 0;
            if (limit) {
                for (int c = 0; c < num_clusters; c++) {
                    int* buf = sdata + c * blockDim.x;
                    buf[tid] += buf[tid + stride];
                }
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        for (int c = 0; c < num_clusters; c++) {
            int* buf = sdata + c * blockDim.x;
            block_counts[block * num_clusters + c] = buf[0];
        }
    }
}

extern "C" __global__ void exclusive_scan_block_counts(
    const int* cluster_offsets,
    const int* block_counts,
    int* block_offsets,
    int num_blocks,
    int num_clusters
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int c = 0; c < num_clusters; c++) {
            int acc = cluster_offsets[c];
            for (int b = 0; b < num_blocks; b++) {
                block_offsets[b * num_clusters + c] = acc;
                acc += block_counts[b * num_clusters + c];
            }
        }
    }
}

extern "C" __global__ void reduce_block_counts(
    const int* block_counts,
    int* cluster_sizes,
    int num_blocks,
    int num_clusters
) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int c = 0; c < num_clusters; c++) {
            int acc = 0;
            for (int b = 0; b < num_blocks; b++) {
                acc += block_counts[b * num_clusters + c];
            }
            cluster_sizes[c] = acc;
        }
    }
}

extern "C" __global__ void parallel_build_cluster_index(
    const int* cluster_assignments,
    const int* block_offsets,
    int* cluster_indices,
    int database_size,
    int num_clusters
) {
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int block = blockIdx.x;
    int base = block * blockDim.x;
    int vec_idx = base + tid;
    __shared__ int s_len;
    if (tid == 0) {
        int rem = database_size - base;
        s_len = rem > blockDim.x ? blockDim.x : (rem > 0 ? rem : 0);
    }
    __syncthreads();
    if (s_len == 0) return;
    int cid = -1;
    if (tid < s_len) cid = cluster_assignments[vec_idx];
    for (int c = 0; c < num_clusters; c++) {
        int* flags = sdata + c * blockDim.x;
        if (tid < s_len) flags[tid] = (cid == c) ? 1 : 0;
        else if (tid < blockDim.x) flags[tid] = 0;
    }
    __syncthreads();
    for (int c = 0; c < num_clusters; c++) {
        int* flags = sdata + c * blockDim.x;
        for (int offset = 1; offset < s_len; offset <<= 1) {
            int v = 0;
            if (tid >= offset && tid < s_len) v = flags[tid - offset];
            __syncthreads();
            if (tid < s_len) flags[tid] += v;
            __syncthreads();
        }
        if (tid < s_len && cid == c) {
            int local_rank = flags[tid] - 1;
            int base_off = block_offsets[block * num_clusters + c];
            cluster_indices[base_off + local_rank] = vec_idx;
        }
        __syncthreads();
    }
}

extern "C" __global__ void cluster_search(
    const float* __restrict__ query_vectors,
    const float* __restrict__ database_vectors,
    const float* __restrict__ cluster_centers,
    const int* __restrict__ cluster_assignments,
    const int* __restrict__ cluster_sizes,
    const int* __restrict__ cluster_indices,
    const int* __restrict__ cluster_offsets,
    int* __restrict__ results,
    int num_queries,
    int database_size,
    int vector_dims,
    int num_clusters
) {
    if (num_queries <= 3000) {
        int query_idx = blockIdx.x;
        if (query_idx >= num_queries) return;

        const float* query = query_vectors + query_idx * vector_dims;

        float cluster_dists[16];
        int cluster_order[16];

        for (int cluster = 0; cluster < num_clusters; cluster++) {
            const float* center = cluster_centers + cluster * vector_dims;
            cluster_dists[cluster] = euclidean_distance(query, center, vector_dims);
            cluster_order[cluster] = cluster;
        }

        int clusters_to_search = (num_queries <= 1000) ? num_clusters :
                                (num_queries <= 2000) ? min(num_clusters, (num_clusters * 3) / 4) :
                                (num_queries <= 2800) ? min(num_clusters, (num_clusters * 2) / 3) :
                                min(num_clusters, max(2, num_clusters / 2));
        if (vector_dims >= 700) {
            int target = max(3, clusters_to_search);
            clusters_to_search = min(num_clusters, target);
        }
        for (int i = 0; i < clusters_to_search; i++) {
            int best = i;
            for (int j = i + 1; j < num_clusters; j++) {
                if (cluster_dists[cluster_order[j]] < cluster_dists[cluster_order[best]]) {
                    best = j;
                }
            }
            int temp = cluster_order[i];
            cluster_order[i] = cluster_order[best];
            cluster_order[best] = temp;
        }

        float min_dist = MAX_FLOAT;
        int best_idx = -1;

        for (int c_idx = 0; c_idx < clusters_to_search; c_idx++) {
            int target_cluster = cluster_order[c_idx];
            if (cluster_sizes[target_cluster] <= 0) continue;

            int start = cluster_offsets[target_cluster];
            int end = start + cluster_sizes[target_cluster];
            for (int p = start; p < end; p++) {
                int vec_idx = cluster_indices[p];
                const float* db_vector = database_vectors + vec_idx * vector_dims;
                float dist = euclidean_distance_bounded(query, db_vector, vector_dims, min_dist);
                if (dist < min_dist) {
                    min_dist = dist;
                    best_idx = vec_idx;
                } else if (vector_dims >= 720 && num_queries <= 5000 && best_idx != -1 && dist <= min_dist + 0.0015f) {
                    float d2 = euclidean_distance_precise_bounded(query, db_vector, vector_dims, min_dist);
                    if (d2 < min_dist) {
                        min_dist = d2;
                        best_idx = vec_idx;
                    }
                }
            }
        }

        if (min_dist == MAX_FLOAT) {
            int base_stride = max(5, database_size / 2000);
            int max_checks = min(database_size / base_stride, 2000);

            for (int phase = 0; phase < 2; phase++) {
                int offset = phase * (base_stride / 2);
                for (int i = 0; i < max_checks / 2; i++) {
                    int db_idx = (offset + i * base_stride) % database_size;

                    const float* db_vector = database_vectors + db_idx * vector_dims;
                    float dist = euclidean_distance_bounded(query, db_vector, vector_dims, min_dist);
                    if (dist < min_dist) {
                        min_dist = dist;
                        best_idx = db_idx;
                    }
                }
            }

            if (best_idx != -1) {
                int radius = min(25, base_stride);
                int start_local = max(0, best_idx - radius);
                int end_local = min(database_size, best_idx + radius + 1);

                for (int i = start_local; i < end_local; i++) {
                    if (i == best_idx) continue;
                    const float* db_vector = database_vectors + i * vector_dims;
                    float dist = euclidean_distance_bounded(query, db_vector, vector_dims, min_dist);
                    if (dist < min_dist) {
                        min_dist = dist;
                        best_idx = i;
                    }
                }
            }
        }

        if (min_dist == MAX_FLOAT) {
            best_idx = 0;
        }

        results[query_idx] = best_idx;
    } else {
        int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (query_idx >= num_queries) return;

        const float* query = query_vectors + query_idx * vector_dims;

        float cluster_dists[16];
        int cluster_order[16];

        for (int cluster = 0; cluster < num_clusters; cluster++) {
            const float* center = cluster_centers + cluster * vector_dims;
            cluster_dists[cluster] = euclidean_distance_high(query, center, vector_dims);
            cluster_order[cluster] = cluster;
        }

        int clusters_to_search = (num_queries <= 3500) ? min(num_clusters, 5) :
                                (num_queries <= 6000) ? min(num_clusters, 4) :
                                (num_queries <= 8000) ? min(num_clusters, 3) :
                                2;
        if (num_queries <= 5000 && vector_dims >= 720) {
            clusters_to_search = num_clusters;
        } else if (vector_dims >= 720) {
            clusters_to_search = num_clusters;
        } else if (vector_dims >= 700) {
            clusters_to_search = max(clusters_to_search, min(num_clusters, (num_clusters * 3) / 4 + 1));
        }
        for (int i = 0; i < clusters_to_search; i++) {
            int best = i;
            for (int j = i + 1; j < num_clusters; j++) {
                if (cluster_dists[cluster_order[j]] < cluster_dists[cluster_order[best]]) {
                    best = j;
                }
            }
            int temp = cluster_order[i];
            cluster_order[i] = cluster_order[best];
            cluster_order[best] = temp;
        }

        float min_dist = MAX_FLOAT;
        int best_idx = -1;

        for (int c_idx = 0; c_idx < clusters_to_search; c_idx++) {
            int target_cluster = cluster_order[c_idx];
            if (cluster_sizes[target_cluster] <= 0) continue;

            int start = cluster_offsets[target_cluster];
            int end = start + cluster_sizes[target_cluster];
            for (int p = start; p < end; p++) {
                int vec_idx = cluster_indices[p];
                const float* db_vector = database_vectors + vec_idx * vector_dims;
                float dist = euclidean_distance_high_bounded(query, db_vector, vector_dims, min_dist);
                if (vector_dims >= 720 && num_queries <= 5000 && dist <= min_dist + 0.0015f) {
                    float d2 = euclidean_distance_precise_bounded(query, db_vector, vector_dims, min_dist);
                    if (d2 < dist) dist = d2;
                }
                if (dist < min_dist) {
                    min_dist = dist;
                    best_idx = vec_idx;
                }
            }
        }

        if (min_dist == MAX_FLOAT) {
            int base_stride = (vector_dims >= 720) ? max(7, database_size / 900) : max(9, database_size / 1200);
            int max_checks = min(database_size / base_stride, (vector_dims >= 720) ? 1600 : 1200);

            for (int phase = 0; phase < 2; phase++) {
                int offset = phase * (base_stride / 3);
                int phase_checks = max_checks / 2;

                for (int i = 0; i < phase_checks; i++) {
                    int db_idx = (offset + i * base_stride) % database_size;

                    const float* db_vector = database_vectors + db_idx * vector_dims;
                    float dist = euclidean_distance_high_bounded(query, db_vector, vector_dims, min_dist);
                    if (dist < min_dist) {
                        min_dist = dist;
                        best_idx = db_idx;
                    }
                }
            }

            if (best_idx != -1) {
                int radius = (vector_dims >= 720) ? min(32, (base_stride * 2) / 3) : min(18, base_stride / 2);
                int start_local = max(0, best_idx - radius);
                int end_local = min(database_size, best_idx + radius + 1);

                for (int i = start_local; i < end_local; i++) {
                    if (i == best_idx) continue;
                    const float* db_vector = database_vectors + i * vector_dims;
                    float dist = euclidean_distance_high_bounded(query, db_vector, vector_dims, min_dist);
                    if (dist < min_dist) {
                        min_dist = dist;
                        best_idx = i;
                    }
                }
            }
        }

        if (min_dist == MAX_FLOAT) {
            best_idx = 0;
        }

        results[query_idx] = best_idx;
    }
}
